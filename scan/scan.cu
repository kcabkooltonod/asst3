#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
__global__ void up_sweep(int* result, int offset, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idx1 = offset * (idx * 2 + 2) - 1;
    int idx2 = offset * (idx * 2 + 1) - 1;
    // if (idx1 < N && idx2 < N)
    result[idx1] += result[idx2];
}

__global__ void clear_last(int* result, int N) {
    result[N - 1] = 0;
}

__global__ void down_sweep(int* result, int offset, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idx1 = offset * (idx * 2 + 2) - 1;
    int idx2 = offset * (idx * 2 + 1) - 1;
    // if (idx1 < N && idx2 < N){
            // printf("idx1: %d idx2: %d\n", idx1, idx2 );

        int t = result[idx2];
        result[idx2] = result[idx1];
        result[idx1] += t;
    // }
}

void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    int blockSize = 512;
    int num_data = N / 2;
    for (int offset = 1; offset <= N /2; offset *= 2) {
        int gridSize = (num_data + blockSize - 1) /blockSize;
        up_sweep<<<gridSize, blockSize>>>(result, offset, N);
        hipDeviceSynchronize();
        num_data /= 2;
    }

    clear_last<<<1, 1>>>(result, N);
    hipDeviceSynchronize();

    num_data = 1;
    for (int offset = N / 2; offset >= 1; offset /= 2) {
        int gridSize = (num_data + blockSize - 1) /blockSize;
        down_sweep<<<gridSize, blockSize>>>(result, offset, N);
        hipDeviceSynchronize();
        num_data *= 2;
    }

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    hipMemset(device_input, 0, rounded_length * sizeof(int));
    hipMemset(device_result, 0, rounded_length * sizeof(int));


    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
__global__ void kernel_cmp (int* input, int* output, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < length - 1){
        output[idx] = input[idx] == input[idx+1] ? 1 : 0;
        // printf("%d, ", output[idx]);
    }
}

__global__ void kernel_sub (int* input, int* output, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < length - 1){
        output[idx] = input[idx+1] - input[idx];
    }
}

__global__ void kernel_collect (int* prefix_sum, int* input, int* output, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < length - 1){
        if(input[idx] == 1){
            output[prefix_sum[idx]] = idx;
        }
    }
}


int find_repeats(int* device_input, int length, int* device_output, int* prefix_sum) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    // int rounded_length = nextPow2(length);
    int *temp = (int*)malloc(sizeof(int)*length);
    int gridSize = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    kernel_cmp<<<gridSize, THREADS_PER_BLOCK>>>(device_input, prefix_sum, length);
    hipDeviceSynchronize();
    exclusive_scan(prefix_sum, length, prefix_sum);
    // hipMemcpy(temp, prefix_sum, sizeof(int) * length, hipMemcpyDeviceToHost);
    kernel_cmp<<<gridSize, THREADS_PER_BLOCK>>>(device_input, device_input, length);
    kernel_collect<<<gridSize, THREADS_PER_BLOCK>>>(prefix_sum, device_input, device_output, length);
    hipDeviceSynchronize();

    int *repeats_num = new int;
    hipMemcpy(repeats_num, prefix_sum +length - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(temp, prefix_sum, sizeof(int) * length, hipMemcpyDeviceToHost);
    // printf("%d\n", repeats_num);
    for(int i = 0; i < length; i++) {
        printf("%d, ",temp[i]);
    }
    return *repeats_num; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int *prefix_sum;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMalloc((void **)&prefix_sum, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output, prefix_sum);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
